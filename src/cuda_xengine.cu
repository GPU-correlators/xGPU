#include "hip/hip_runtime.h"
/*
  Simple cross-product, outputs in correct triangular form.

  - Coalescing memory access in all reads
  - No memory coalscing in writes (will be fixed)
  - Shared memory reads of type float2 to reduce global memory traffic
  - Each thread works on a 2x2 tile of data

  On a GTX 480 with >= 512 tiles this kernel achieve in excess of a
  teraflop.
 */

#include <stdio.h>
#include <unistd.h>
#include <stdint.h>

#include "xgpu.h"
#include "xgpu_info.h"
#include "xgpu_version.h"
#include "cube/cube.h"

// whether we are writing the matrix back to device memory (used for benchmarking)
static int writeMatrix = 1;
// this must be enabled for this option to work though, slightly hurts performance
//#define WRITE_OPTION 

// System page size (used for rounding size passed to hipHostRegister)
static long page_size = sysconf(_SC_PAGE_SIZE);

typedef struct XGPUInternalContextStruct {
  // Which device this context applies to
  int device;

  //memory pointers on the device
  ComplexInput *array_d[2];
  Complex *matrix_d;

  // used for overlapping comms and compute
  hipStream_t streams[2];
  hipEvent_t copyCompletion[2];
  hipEvent_t kernelCompletion[2];

  // texture channel descriptor
  hipChannelFormatDesc channelDesc;

  // Host input array that we allocated and should free
  ComplexInput * free_array_h;

  // Host input array that we registered and should unregister
  ComplexInput * unregister_array_h;

  // Whether xgpuSetHostInputBuffer has been called
  bool array_h_set;
  bool register_host_array;

  // Host output array that we allocated and should free
  Complex * free_matrix_h;

  // Host output array that we registered and should unregister
  Complex * unregister_matrix_h;

  // Whether xgpuSetHostOutputBuffer has been called
  bool matrix_h_set;
  bool register_host_matrix;
} XGPUInternalContext;

#define TILE_HEIGHT 8
#define TILE_WIDTH 8
#define NPOL 2

#define REG_TILE_NBASELINE ((NSTATION/2+1)*(NSTATION/4))

#ifndef FIXED_POINT
// texture declaration for FP32 reads
static texture<float2, 1, hipReadModeElementType> tex1dfloat2;
static texture<float2, 2, hipReadModeElementType> tex2dfloat2;
#else
#ifdef DP4A
// texture declaration for swizzled 8-bit fixed point reads
static texture<int2, 1, hipReadModeElementType> tex1dchar4;
static texture<int2, 2, hipReadModeElementType> tex2dchar4;
#else
// texture declaration for 8-bit fixed point reads
static texture<char2, 1, hipReadModeNormalizedFloat> tex1dfloat2;
static texture<char2, 2, hipReadModeNormalizedFloat> tex2dfloat2;
#endif
#endif

// array holding indices for which matrix we are doing the output to at a given iteration
#if (NPULSAR > 0)
static __device__ __constant__ unsigned char tIndex[PIPE_LENGTH*NFREQUENCY];
#endif

#define checkCudaError() do {                           \
    hipError_t error = hipGetLastError();		\
    if (error != hipSuccess) {				\
      fprintf(stderr, "(CUDA) %s", hipGetErrorString(error));	\
      fprintf(stderr, " (" __FILE__ ":%d)\n", __LINE__);		\
      return XGPU_CUDA_ERROR;						\
    }							\
  } while (0)

#ifdef TIME_CUDA_CALLS
#define CLOCK_GETTIME(clk_id, tp) clock_gettime(clk_id, tp)
#define PRINT_ELAPASED(f,t) printf("%s %ld ns\n", f, t)
#else
#define CLOCK_GETTIME(clk_id, tp)
#define PRINT_ELAPASED(f,t)
#endif

#include "kernel.cuh"

static XGPUInfo compiletime_info = {
  .npol =        NPOL,
  .nstation =    NSTATION,
  .nbaseline =   NBASELINE,
  .nfrequency =  NFREQUENCY,
  .ntime =       NTIME,
  .ntimepipe =   NTIME_PIPE,
#ifdef FIXED_POINT
  .input_type =  XGPU_INT8,
#else
  .input_type =  XGPU_FLOAT32,
#endif
#ifdef DP4A
  .compute_type = XGPU_INT8,
#else
  .compute_type = XGPU_FLOAT32,
#endif
  .vecLength  =  NFREQUENCY * NTIME * NSTATION * NPOL,
  .vecLengthPipe = NFREQUENCY * NTIME_PIPE * NSTATION * NPOL,
#if (MATRIX_ORDER == REGISTER_TILE_TRIANGULAR_ORDER)
  .matLength =   NFREQUENCY * ((NSTATION/2+1)*(NSTATION/4)*NPOL*NPOL*4) * (NPULSAR + 1),
#else
  // Matrix length is same for REAL_IMAG_TRIANGULAR_ORDER and TRIANGULAR_ORDER
  .matLength =   NFREQUENCY * ((NSTATION+1)*(NSTATION/2)*NPOL*NPOL) * (NPULSAR + 1),
#endif
  .triLength =   NFREQUENCY * ((NSTATION+1)*(NSTATION/2)*NPOL*NPOL) * (NPULSAR + 1),
  .matrix_order = MATRIX_ORDER,
  .shared_atomic_size = SHARED_ATOMIC_SIZE,
  .complex_block_size = COMPLEX_BLOCK_SIZE
};

// This stringification trick is from "info cpp"
#define STRINGIFY1(s) #s
#define STRINGIFY(s) STRINGIFY1(s)
static const char xgpu_version[] = STRINGIFY(XGPU_VERSION);

const char * xgpuVersionString()
{
  return xgpu_version;
}

// Populate XGPUInfo structure with compile-time parameters.
void xgpuInfo(XGPUInfo *pcxs)
{
  pcxs->npol           = compiletime_info.npol;
  pcxs->nstation       = compiletime_info.nstation;
  pcxs->nbaseline      = compiletime_info.nbaseline;
  pcxs->nfrequency     = compiletime_info.nfrequency;
  pcxs->ntime          = compiletime_info.ntime;
  pcxs->ntimepipe      = compiletime_info.ntimepipe;
  pcxs->input_type     = compiletime_info.input_type;
  pcxs->compute_type   = compiletime_info.compute_type;
  pcxs->vecLength      = compiletime_info.vecLength;
  pcxs->vecLengthPipe  = compiletime_info.vecLengthPipe;
  pcxs->matLength      = compiletime_info.matLength;
  pcxs->triLength      = compiletime_info.triLength;
  pcxs->matrix_order   = compiletime_info.matrix_order;
  pcxs->shared_atomic_size = compiletime_info.shared_atomic_size;
  pcxs->complex_block_size = compiletime_info.complex_block_size;
}

// Initialize the XGPU.  The device number is intentionally not part of the
// context because the device number needs to be maintained as part of the
// internal context (.e.g to ensure consistency with the device on which memory
// was allocated).
//
// TODO Cleanup as needed if returning due to error
int xgpuInit(XGPUContext *context, int device_flags)
{
  int error = XGPU_OK;

  CUBE_INIT();

  // Allocate internal context
  XGPUInternalContext *internal = (XGPUInternalContext *)malloc(sizeof(XGPUInternalContext));
  if(!internal) {
    // Uh-oh!
    return XGPU_OUT_OF_MEMORY;
  }
  context->internal = internal;
  internal->device = device_flags & XGPU_DEVICE_MASK;
  internal->array_h_set  = false;
  internal->matrix_h_set = false;
  internal->register_host_array  = true;
  internal->register_host_matrix = true;
  if( device_flags & XGPU_DONT_REGISTER_ARRAY ) {
	  internal->register_host_array = false;
  }
  if( device_flags & XGPU_DONT_REGISTER_MATRIX ) {
	  internal->register_host_matrix = false;
  }

  long long unsigned int vecLengthPipe = compiletime_info.vecLengthPipe;
  long long unsigned int matLength = compiletime_info.matLength;

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    printf("No CUDA devices found");
    exit(-1);
  }

  hipDeviceProp_t deviceProp;
  for(int i=0; i<deviceCount; i++) {
    hipGetDeviceProperties(&deviceProp, i);
    printf("Found device %d: %s\n", i, deviceProp.name);
  }

  hipGetDeviceProperties(&deviceProp, internal->device);
  printf("Using device %d: %s\n", internal->device, deviceProp.name);

  //assign the device
  hipSetDevice(internal->device);
  checkCudaError();

  // Setup input buffer
  internal->unregister_array_h = NULL;
  internal->free_array_h = NULL;
  if( internal->register_host_array ) {
	  // TODO error check
	  xgpuSetHostInputBuffer(context);
  }

  // Setup output buffer
  internal->unregister_matrix_h = NULL;
  internal->free_matrix_h = NULL;
  if( internal->register_host_matrix ) {
	  // TODO error check
	  xgpuSetHostOutputBuffer(context);
  }

  //allocate memory on device
  hipMalloc((void **) &(internal->array_d[0]), vecLengthPipe*sizeof(ComplexInput));
  hipMalloc((void **) &(internal->array_d[1]), vecLengthPipe*sizeof(ComplexInput));
  hipMalloc((void **) &(internal->matrix_d), matLength*sizeof(Complex));
  checkCudaError();
  
  //clear out any previous values
  hipMemset(internal->array_d[0], '\0', vecLengthPipe*sizeof(ComplexInput));
  hipMemset(internal->array_d[1], '\0', vecLengthPipe*sizeof(ComplexInput));
  checkCudaError();

  // Clear device integration bufer
  error = xgpuClearDeviceIntegrationBuffer(context);
  if(error != XGPU_OK) {
    return error;
  }

  // create the streams
  for(int i=0; i<2; i++) hipStreamCreate(&(internal->streams[i]));
  checkCudaError();

  // create the events
  for (int i=0; i<2; i++) {
    hipEventCreateWithFlags(&(internal->kernelCompletion[i]), hipEventDisableTiming);
    hipEventCreateWithFlags(&(internal->copyCompletion[i]), hipEventDisableTiming);
  }
  checkCudaError();

#ifndef FIXED_POINT
  internal->channelDesc = hipCreateChannelDesc<float2>();
#else
#ifdef DP4A
  internal->channelDesc = hipCreateChannelDesc<int2>();
#else
  internal->channelDesc = hipCreateChannelDesc<char2>();
#endif // DP4A
#endif // FIXED_POINT

#if NPULSAR > 0
  unsigned char timeIndex[PIPE_LENGTH*NFREQUENCY];
  for (int tf=0; tf<PIPE_LENGTH*NFREQUENCY; tf++) timeIndex[tf] = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(tIndex), timeIndex, PIPE_LENGTH*NFREQUENCY*sizeof(unsigned char), hipMemcpyHostToDevice);

  checkCudaError();

  // check symbols are copied over
  unsigned char timeIndex2[PIPE_LENGTH*NFREQUENCY];
  hipMemcpyFromSymbol(timeIndex2[t], HIP_SYMBOL(tIndex[t]), PIPE_LENGTH*NFREQUENCY*sizeof(unsigned char), hipMemcpyDeviceToHost);  
  for (int tf=0; tf<PIPE_LENGTH*NFREQUENCY; tf++) {
    for (int f=0; f<NFREQUENCY; f++) 
      if (timeIndex[t][f] != timeIndex2[t][f]) 
	fprintf(stderr, "Index copy failed: t = %d, f = %d, original = %d, copy = %d\n", 
	       t, f, timeIndex[t][f], timeIndex2[t][f]);
  }
#endif

  // check whether texture dimensions are ok
#if TEXTURE_DIM == 2
#ifdef DP4A
  if((NFREQUENCY * NSTATION * NPOL > deviceProp.maxTexture2DLinear[0]) ||
     (NTIME_PIPE/4 > deviceProp.maxTexture2DLinear[1])) {
    return XGPU_INSUFFICIENT_TEXTURE_MEMORY;
  }
#else
  if((NFREQUENCY * NSTATION * NPOL > deviceProp.maxTexture2DLinear[0]) ||
     (NTIME_PIPE > deviceProp.maxTexture2DLinear[1])) {
    return XGPU_INSUFFICIENT_TEXTURE_MEMORY;
  }
#endif
#elif TEXTURE_DIM == 1
  // Surprisingly, this appears not to be a problem with 1D textures.  On a
  // GeForce GTX 580 (i.e. Fermi device), deviceQuery returns 65536 as
  // maxTexture1D, yet the default sizes use 10 * 256 * 2 * 100 * 2 == 1024000
  // bytes of 1D texture without any problems.  Perhaps the value of
  // maxTexture1D returned by hipGetDeviceProperties is wrong?
#ifdef DP4A
  if (NFREQUENCY * NSTATION * NPOL * (NTIME_PIPE/4) > deviceProp.maxTexture1DLinear) {
    return XGPU_INSUFFICIENT_TEXTURE_MEMORY;
  }
#else
  if (NFREQUENCY * NSTATION * NPOL * NTIME_PIPE > deviceProp.maxTexture1DLinear) {
    return XGPU_INSUFFICIENT_TEXTURE_MEMORY;
  }
#endif
#endif 

  return XGPU_OK;
}

// Clear the device integration buffer
int xgpuClearDeviceIntegrationBuffer(XGPUContext *context)
{
  long long unsigned int matLength = compiletime_info.matLength;

  XGPUInternalContext *internal = (XGPUInternalContext *)context->internal;
  if(!internal) {
    return XGPU_NOT_INITIALIZED;
  }
  //assign the device
  hipSetDevice(internal->device);

  hipMemset(internal->matrix_d, '\0', matLength*sizeof(Complex));
  checkCudaError();
  return XGPU_OK;
}

#define ELAPSED_NS(start,stop) \
  (((int64_t)stop.tv_sec-start.tv_sec)*1000*1000*1000+(stop.tv_nsec-start.tv_nsec))

// Specify a new host input buffer.
int xgpuSetHostInputBuffer(XGPUContext *context)
{
#ifdef TIME_CUDA_CALLS
  struct timespec a, b;
#endif

  XGPUInternalContext *internal = (XGPUInternalContext *)context->internal;
  if(!internal) {
    return XGPU_NOT_INITIALIZED;
  }

  internal->array_h_set = true;

  //assign the device
  CLOCK_GETTIME(CLOCK_MONOTONIC, &a);
  hipSetDevice(internal->device);
  CLOCK_GETTIME(CLOCK_MONOTONIC, &b);
  PRINT_ELAPASED("hipSetDevice", ELAPSED_NS(a,b));

  if(internal->free_array_h) {
    CLOCK_GETTIME(CLOCK_MONOTONIC, &a);
    hipHostFree(internal->free_array_h);
    CLOCK_GETTIME(CLOCK_MONOTONIC, &b);
    PRINT_ELAPASED("hipHostFree", ELAPSED_NS(a,b));
  }
  if(internal->unregister_array_h) {
    CLOCK_GETTIME(CLOCK_MONOTONIC, &a);
    hipHostUnregister(internal->unregister_array_h);
    CLOCK_GETTIME(CLOCK_MONOTONIC, &b);
    PRINT_ELAPASED("hipHostUnregister", ELAPSED_NS(a,b));
  }

  if(context->array_h) {
    if( internal->register_host_array ) {
      // Register caller-allocated host memory with CUDA.
      // Round address down to nearest page_size boundary
      uintptr_t ptr_in = (uintptr_t)context->array_h;
      uintptr_t ptr_aligned = ptr_in - (ptr_in % page_size);
      // Compute length starting with compile time requirement
      size_t length = context->array_len * sizeof(ComplexInput);
      // TODO Verify that length is at least
      // "compiletime_info.vecLength*sizeof(ComplexInput)"

      // Add in any rounding that was done to the input pointer
      length += (ptr_in - ptr_aligned);
      // Round length up to next multiple of page size
      length = (length+page_size-1) / page_size * page_size;
#ifdef VERBOSE
      fprintf(stderr, "page aligned context->array_h = %p\n", ptr_aligned);
      fprintf(stderr, "length = %lx\n", length);
#endif
      CLOCK_GETTIME(CLOCK_MONOTONIC, &a);
      hipHostRegister((void *)ptr_aligned, length, 0);
      CLOCK_GETTIME(CLOCK_MONOTONIC, &b);
      PRINT_ELAPASED("hipHostRegister", ELAPSED_NS(a,b));
      internal->unregister_array_h = (ComplexInput *)ptr_aligned;
      internal->free_array_h = NULL;
      checkCudaError();
    }
    else {
      internal->unregister_array_h = NULL;
      internal->free_array_h = NULL;
    }
  } else {
    // allocate host memory
    context->array_len = compiletime_info.vecLength;
    CLOCK_GETTIME(CLOCK_MONOTONIC, &a);
    hipHostMalloc(&(context->array_h), context->array_len*sizeof(ComplexInput));
    CLOCK_GETTIME(CLOCK_MONOTONIC, &b);
    PRINT_ELAPASED("hipHostMalloc", ELAPSED_NS(a,b));
    internal->free_array_h = context->array_h;
    internal->unregister_array_h = NULL;
    checkCudaError();
  }

  // Init input_offset to 0
  context->input_offset = 0;

  return XGPU_OK;
}

// Specify a new host output buffer.
int xgpuSetHostOutputBuffer(XGPUContext *context)
{
  XGPUInternalContext *internal = (XGPUInternalContext *)context->internal;
  if(!internal) {
    return XGPU_NOT_INITIALIZED;
  }

  internal->matrix_h_set = true;

  //assign the device
  hipSetDevice(internal->device);

  if(internal->free_matrix_h) {
    hipHostFree(internal->free_matrix_h);
  }
  if(internal->unregister_matrix_h) {
    hipHostUnregister(internal->unregister_matrix_h);
  }

  if(context->matrix_h) {
    if( internal->register_host_matrix ) {
      // Register caller-allocated host memory with CUDA.
      // This requires that the caller allocated the memory properly vis-a-vis
      // the requirements of hipHostRegister!
      // Round address down to nearest page_size boundary
      uintptr_t ptr_in = (uintptr_t)context->matrix_h;
      uintptr_t ptr_aligned = ptr_in - (ptr_in % page_size);
      // Compute length starting with compile time requirement
      size_t length = context->matrix_len * sizeof(Complex);
      // TODO Verify that length is at least
      // "compiletime_info.matLength*sizeof(Complex)"

      // Add in any rounding that was done to the input pointer
      length += (ptr_in - ptr_aligned);
      // Round length up to next multiple of page size
      length = (length+page_size-1) / page_size * page_size;
#ifdef VERBOSE
      fprintf(stderr, "page aligned context->matrix_h = %p\n", ptr_aligned);
      fprintf(stderr, "length = %lx\n", length);
#endif
      hipHostRegister((void *)ptr_aligned, length, 0);
      internal->unregister_matrix_h = (Complex *)ptr_aligned;
      internal->free_matrix_h = NULL;
      checkCudaError();
    }
    else {
      internal->unregister_matrix_h = NULL;
      internal->free_matrix_h = NULL;
    }
  } else {
    // allocate host memory
    context->matrix_len = compiletime_info.matLength;
    hipHostMalloc(&(context->matrix_h), context->matrix_len*sizeof(Complex));
    internal->free_matrix_h = context->matrix_h;
    internal->unregister_matrix_h = NULL;
    checkCudaError();
  }

  // Init output_offset to 0
  context->output_offset = 0;

  return XGPU_OK;
}

// Free up the memory on the host and device
void xgpuFree(XGPUContext *context)
{
  XGPUInternalContext *internal = (XGPUInternalContext *)context->internal;

  if(internal) {
    //assign the device
    hipSetDevice(internal->device);

    for(int i=0; i<2; i++) {
      hipStreamDestroy(internal->streams[i]);
      hipEventDestroy(internal->copyCompletion[i]);
      hipEventDestroy(internal->kernelCompletion[i]);
    }

    if(internal->free_array_h) {
      hipHostFree(internal->free_array_h);
      context->array_h = NULL;
    }
    if(internal->unregister_array_h) {
      hipHostUnregister(internal->unregister_array_h);
      context->array_h = NULL;
    }
    if(internal->free_matrix_h) {
      hipHostFree(internal->free_matrix_h);
      context->matrix_h = NULL;
    }
    if(internal->unregister_matrix_h) {
      hipHostUnregister(internal->unregister_matrix_h);
      context->matrix_h = NULL;
    }

    hipFree(internal->array_d[1]);
    hipFree(internal->array_d[0]);
    hipFree(internal->matrix_d);

    free(internal);
    context->internal = NULL;
  }

  CUBE_WRITE();
}

int xgpuCudaXengine(XGPUContext *context, int syncOp)
{
  XGPUInternalContext *internal = (XGPUInternalContext *)context->internal;
  if(!internal) {
    return XGPU_NOT_INITIALIZED;
  }

  // xgpuSetHostInputBuffer and xgpuSetHostOutputBuffer must have been called
  if( !internal->array_h_set || !internal->matrix_h_set ) {
    return XGPU_HOST_BUFFER_NOT_SET;
  }

  //assign the device
  hipSetDevice(internal->device);

  ComplexInput **array_d = internal->array_d;
  hipStream_t *streams = internal->streams;
  hipEvent_t *copyCompletion = internal->copyCompletion;
  hipEvent_t *kernelCompletion = internal->kernelCompletion;
  hipChannelFormatDesc channelDesc = internal->channelDesc;

  // set pointers to the real and imaginary components of the device matrix
#ifndef DP4A
  float4 *matrix_real_d = (float4 *)(internal->matrix_d);
  float4 *matrix_imag_d = (float4 *)(internal->matrix_d + compiletime_info.matLength/2);
#else
  int4 *matrix_real_d = (int4 *)(internal->matrix_d);
  int4 *matrix_imag_d = (int4 *)(internal->matrix_d + compiletime_info.matLength/2);
#endif

  int Nblock = compiletime_info.nstation/min(TILE_HEIGHT,TILE_WIDTH);
  ComplexInput *array_load;
  ComplexInput *array_compute; 

  dim3 dimBlock(TILE_WIDTH,TILE_HEIGHT,1);
  //allocated exactly as many thread blocks as are needed
  dim3 dimGrid(((Nblock/2+1)*(Nblock/2))/2, compiletime_info.nfrequency);

  CUBE_ASYNC_START(ENTIRE_PIPELINE);

  // Need to fill pipeline before loop
  long long unsigned int vecLengthPipe = compiletime_info.vecLengthPipe;
  ComplexInput *array_hp = context->array_h + context->input_offset;
  // Only start the transfer once the kernel has completed processing input
  // buffer 0.  This is a no-op unless previous call to xgpuCudaXengine() had
  // SYNCOP_NONE or SYNCOP_SYNC_TRANSFER.
  hipStreamWaitEvent(streams[0], kernelCompletion[0], 0);
  CUBE_ASYNC_COPY_CALL(array_d[0], array_hp, vecLengthPipe*sizeof(ComplexInput), hipMemcpyHostToDevice, streams[0]);
  hipEventRecord(copyCompletion[0], streams[0]); // record the completion of the h2d transfer
  checkCudaError();

  CUBE_ASYNC_START(PIPELINE_LOOP);

#ifdef POWER_LOOP
  for (int q=0; ; q++)
#endif
  for (int p=1; p<PIPE_LENGTH; p++) {
    array_compute = array_d[(p+1)%2];
    array_load = array_d[p%2];

    // Kernel Calculation
#if TEXTURE_DIM == 2
#ifndef DP4A
    hipBindTexture2D(0, tex2dfloat2, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL, NTIME_PIPE,
		      NFREQUENCY*NSTATION*NPOL*sizeof(ComplexInput));
#else
    hipBindTexture2D(0, tex2dchar4, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL, NTIME_PIPE/4,
		      NFREQUENCY*NSTATION*NPOL*2*sizeof(char4));
#endif
#else
#ifndef DP4A
    hipBindTexture(0, tex1dfloat2, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL*NTIME_PIPE*sizeof(ComplexInput));
#else
    hipBindTexture(0, tex1dchar4, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL*(NTIME_PIPE/4)*sizeof(int2));
#endif
#endif
    hipStreamWaitEvent(streams[1], copyCompletion[(p+1)%2], 0); // only start the kernel once the h2d transfer is complete
    CUBE_ASYNC_KERNEL_CALL(shared2x2, dimGrid, dimBlock, 0, streams[1], 
			   matrix_real_d, matrix_imag_d, NSTATION, writeMatrix);
    hipEventRecord(kernelCompletion[(p+1)%2], streams[1]); // record the completion of the kernel
    checkCudaError();

    // Download next chunk of input data
    hipStreamWaitEvent(streams[0], kernelCompletion[p%2], 0); // only start the transfer once the kernel has completed
    CUBE_ASYNC_COPY_CALL(array_load, array_hp+p*vecLengthPipe, vecLengthPipe*sizeof(ComplexInput), hipMemcpyHostToDevice, streams[0]);
    hipEventRecord(copyCompletion[p%2], streams[0]); // record the completion of the h2d transfer
    checkCudaError();
  }

  CUBE_ASYNC_END(PIPELINE_LOOP);

  array_compute = array_d[(PIPE_LENGTH+1)%2];
  // Final kernel calculation
#if TEXTURE_DIM == 2
#ifndef DP4A
    hipBindTexture2D(0, tex2dfloat2, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL, NTIME_PIPE,
		      NFREQUENCY*NSTATION*NPOL*sizeof(ComplexInput));
#else
    hipBindTexture2D(0, tex2dchar4, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL, NTIME_PIPE/4,
		      NFREQUENCY*NSTATION*NPOL*2*sizeof(char4));
#endif
#else
#ifndef DP4A
    hipBindTexture(0, tex1dfloat2, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL*NTIME_PIPE*sizeof(ComplexInput));
#else
    hipBindTexture(0, tex1dchar4, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL*(NTIME_PIPE/4)*sizeof(int2));
#endif
#endif
  hipStreamWaitEvent(streams[1], copyCompletion[(PIPE_LENGTH+1)%2], 0);
  CUBE_ASYNC_KERNEL_CALL(shared2x2, dimGrid, dimBlock, 0, streams[1], matrix_real_d, matrix_imag_d,
			 NSTATION, writeMatrix);

  if(syncOp == SYNCOP_DUMP) {
    checkCudaError();
    //copy the data back, employing a similar strategy as above
    CUBE_COPY_CALL(context->matrix_h + context->output_offset, internal->matrix_d, compiletime_info.matLength*sizeof(Complex), hipMemcpyDeviceToHost);
    checkCudaError();
  } else if(syncOp == SYNCOP_SYNC_COMPUTE) {
    // Synchronize on the compute stream (i.e. wait for it to complete)
    hipStreamSynchronize(streams[1]);
  } else {
      // record the completion of the kernel for next call
      hipEventRecord(kernelCompletion[(PIPE_LENGTH+1)%2], streams[1]);
      checkCudaError();

      if(syncOp == SYNCOP_SYNC_TRANSFER) {
        // Synchronize on the transfer stream (i.e. wait for it to complete)
        hipStreamSynchronize(streams[0]);
      }
  }

  CUBE_ASYNC_END(ENTIRE_PIPELINE);

  return XGPU_OK;
}
